#include "hip/hip_runtime.h"
#include "raycast_cuda.h"
#include "platform.h"

#include "framebuffer.h"
#include "volume.h"
#include "camera.h"

#include "voxel.h"
#include <stdio.h>
#include <stdint.h>
#include <sstream>
#include <glm/mat4x4.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtx/transform.hpp>
#include <glm/gtc/random.hpp>
#include <hip/hip_vector_types.h>
#include "bmp.h"
#include <fstream>

// Define global volume and framebuffer handles, for now at least
texture<uint32_t,3> volume_texture;
texture<float4,1> ssao_kernel;
texture<float4,1> ssao_noise;
surface<void, 2> fb_color_surface;
surface<void, 2> fb_normal_depth_surface;

VOXOWL_HOST
float
random( float min, float max ) {
    return min + static_cast <float> (rand()) /( static_cast <float> (RAND_MAX/(max-min)));
}

// Simple return-on-error mechanism to improve readability
#define RETURN_IF_ERR(err) { if( setCudaErrorStr((err), __FILE__, __LINE__) ) return false; }
inline bool
RaycasterCUDA::setCudaErrorStr(hipError_t code, char *file, int line )
{
    if (code != hipSuccess) 
    {
        std::stringstream s;
        s <<  "Error (cuda): '" 
          << hipGetErrorString(code) 
          <<  "' in "
          << file 
          << " at line "
          << line;

        setError( true, s.str() );
        return true;
    }
    setError( false, std::string() );
    return false;
}

VOXOWL_DEVICE
glm::vec4
voxel( volumeDevice_t v, glm::ivec3 index ) {
    glm::ivec3 block =glm_ivec3_16( blockPosition( v.format, ivec3_16( index ) ) );
    glm::vec4 vox;

    switch( v.format ) {
        case VOXEL_RGBA_UINT32: {
            vox =unpackRGBA_UINT32( tex3D( volume_texture, block.x, block.y, block.z ) );
            break;
        }
        case VOXEL_INTENSITY_UINT8: {
            uint8_t gray =tex3D( volume_texture, block.x, block.y, block.z );
            vox =glm::vec4( unpackINTENSITY_UINT8( gray ) );
            vox.a =(float)(vox.r != 0.f);
            break;
        }
        case VOXEL_BITMAP_UINT8: {
            uint8_t bitmap =tex3D( volume_texture, block.x, block.y, block.z );
            int bit_offs =index.z % voxelsPerBlock( v.format );
            //int bit_offs =index.z - block.z*8;
            vox =glm::vec4( (int)unpackBIT_UINT8( bitmap, bit_offs ) );
            break;
        }
        case VOXEL_RGB24_8ALPHA1_UINT32: {
            uint32_t rgb24_8alpha1 =tex3D( volume_texture, block.x, block.y, block.z );
            int bit_offs =index.z % voxelsPerBlock( v.format );
            vox =unpackRGBA_RGB24_8ALPHA1_UINT32( rgb24_8alpha1, bit_offs );

            break;
        }
        case VOXEL_RGB24A1_UINT32: {
            vox =unpackRGB24A1_UINT32( tex3D( volume_texture, block.x, block.y, block.z ) );
            break;
        }
    }
    vox.a *= (float)( glm::all( glm::greaterThanEqual( index, glm::ivec3(0) ) ) && glm::all( glm::lessThan( index, v.size ) ) );

    return vox;
}

/* Cast one ray r into the volume bounded by v. The actual volume data is obtained from the global volume texture */
VOXOWL_DEVICE
fragment_t
raycast( volumeDevice_t v, const ray_t& r ) {
    double tmin, tmax;
    glm::ivec3 size =v.size;
    
    fragment_t frag;
    frag.color =glm::vec4( 0,0,0,1 );
    frag.normal =glm::vec3(0);
    frag.position =glm::vec3(0);
    frag.position_vs =glm::vec3(0);

    box_t b = volumeSizeToAABB( size );
    if( !rayAABBIntersect( r, b, tmin, tmax ) )
        return frag;

    glm::vec3 rayEntry = r.origin + r.direction * (float)max( 0.0, tmin );
    glm::vec3 rayExit = r.origin + r.direction * (float)tmax;

    // Determine the side of the unit cube the ray enters
    // In order to do this, we need the component with the largest absolute number
    // These lines are optimized to do so without branching
    const glm::ivec3 box_plane( 0, 1, 2 ); // X, Y and Z dividing planes
    glm::vec3 r0 = glm::abs( rayEntry / b.max );
    float largest =max( r0.x, max( r0.y, r0.z ) ); // Largest relative component
    glm::ivec3 r1 = glm::floor( r0 / largest ); // Vector with a '1' at the largest component
    int side = glm::clamp( glm::dot( box_plane, r1 ), 0, 2 );
   
    // Map the ray entry from unit-cube space to voxel space
    largest =max( size.x, max( size.y, size.z ) );
    glm::vec3 rayEntry_vs =(rayEntry + b.max) * largest;

    // Calculate the index in the volume by chopping off the decimal part
    glm::ivec3 index = glm::clamp( 
        glm::ivec3( glm::floor( rayEntry_vs ) ) ,
        glm::ivec3( 0 ),
        glm::ivec3( size.x-1, size.y-1, size.z-1 ) );

    frag.position_vs = glm::clamp( 
        glm::vec3( rayEntry_vs ) ,
        glm::vec3( 0 ),
        glm::vec3( size.x-1, size.y-1, size.z-1 ) );

    // Determine the sign of the stepping through the volume
    glm::ivec3 step = glm::sign( r.direction );

    // deltaDist gives the distance on the ray path for each following dividing plane
    glm::vec3 deltaDist =glm::abs( glm::vec3( glm::length( r.direction ) ) / r.direction );

    // Computes the distances to the next voxel for each component
    glm::vec3 boxDist = ( sign( r.direction ) * (glm::vec3(index) - rayEntry_vs)
                        + (sign( r.direction ) * 0.5f ) + 0.5f ) * deltaDist;

    while(1) {

        if( index[side] < 0 || index[side] >= size[side] )
            break;
        
        glm::vec4 vox = voxel( v, glm::ivec3( glm::floor( frag.position_vs ) ) );
/*        vox.r *= (3-side)/3.f;
        vox.g *= (3-side)/3.f;
        vox.b *= (3-side)/3.f;*/

        frag.color =blendF2B( vox, frag.color );

        if( vox.a == 1.f ) {
            // We calculate the position in unit-cube space..
            frag.position =frag.position_vs / (float)largest - b.max;
            // ..and the normal of the current 'face' of the voxel
            frag.normal[side] = -step[side];
            break;
        }


        // Branchless equivalent for
        //for( int i =0; i < 3; i++ ) 
        //    if( boxDist[side] > boxDist[i] )
        //        side =i;*/
        glm::bvec3 b0= glm::lessThan( boxDist, glm::vec3( boxDist.y, boxDist.z, boxDist.x ) /*boxDist.yzx()*/ );
        glm::bvec3 b1= glm::lessThanEqual( boxDist, glm::vec3( boxDist.z, boxDist.x, boxDist.y ) /*boxDist.zxy()*/ );
        glm::ivec3 mask =glm::ivec3( b0.x && b1.x, b0.y && b1.y, b0.z && b1.z );
        side = glm::dot( box_plane, mask );

        boxDist[side] += deltaDist[side];
        index[side] += step[side];
        frag.position_vs[side] += step[side];
    }

    frag.color.a = 1.f - frag.color.a;

    return frag;
}

/* Parallel raycast kernel. Computes one fragment depending on position in the threadblock and writes in to the framebuffer */
VOXOWL_CUDA_KERNEL
void
computeFragment( raycastInfo_t raycast_info, volumeDevice_t volume, framebufferDevice_t framebuffer, ssaoInfo_t ssao_info, glm::mat4 mat_projection ) {
    // Calculate screen coordinates
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Next, we calculate the ray vector based on the screen coordinates
    glm::vec3 leftNormal = raycast_info.upperLeftNormal;
    glm::vec3 rightNormal = raycast_info.upperRightNormal;
       
    ray_t r;
    r.origin =raycast_info.origin;
    leftNormal += raycast_info.leftNormalYDelta * (float)y;
    rightNormal += raycast_info.rightNormalYDelta * (float)y;
    r.direction = leftNormal;
    
    glm::vec3 normalXDelta = (rightNormal - leftNormal) * raycast_info.invWidth;
    r.direction +=normalXDelta * (float)x;

    // Initialize variables used during sampling
    const float inv_aa_samples =1.f / (float)(framebuffer.aaXSamples * framebuffer.aaYSamples);
    fragment_t frag;
    frag.color =glm::vec4(0);
    frag.position =glm::vec3(0);
    frag.position_vs =glm::vec3(0);
    frag.normal =glm::vec3(0);

    for( int i =0; i < framebuffer.aaXSamples; i++ )
        for( int j =0; j < framebuffer.aaYSamples; j++ ) {

            // Shift the ray direction depending on the AA sample
            glm::vec3 raydir = r.direction 
                + (float)i /*/ ( 1 * framebuffer.aaXSamples)*/ * normalXDelta
                + (float)j /*/ ( 1 * framebuffer.aaYSamples)*/ * raycast_info.leftNormalYDelta;
    
            // Transform the ray from world-space to unit-cube-space
            ray_t r_cube;
            r_cube.direction =glm::normalize( glm::mat3( raycast_info.matInvModelView ) * raydir );
            r_cube.origin =r.origin;
            
            // Cast the ray and average it with the other samples
            fragment_t f = raycast( volume, r_cube );
            frag.color += f.color;
            frag.position += f.position;
            frag.position_vs +=f.position_vs;
            frag.normal +=  f.normal;

        }

    // Average out
    frag.color *=inv_aa_samples;
    frag.position *=inv_aa_samples;
    frag.position_vs *=inv_aa_samples;
    frag.normal *=inv_aa_samples;

    // VSAO
    float4 noise =tex1D( ssao_noise, (float)x*y );
    glm::vec3 rvec( noise.x, noise.y, noise.z );

    // Setup the TBN matrix
    glm::vec3 tangent = glm::normalize(rvec - frag.normal * glm::dot(rvec, frag.normal));
    glm::vec3 bitangent = glm::cross(frag.normal, tangent);
    glm::mat3 tbn(tangent, bitangent, frag.normal);

    // Obtain the samples
    float occlusion =0.f;
    for (int i = 0; i < ssao_info.kernelSize; ++i) {
        // get sample position:

        float4 k =tex1D( ssao_kernel, i );
        glm::vec3 kernel_i( k.x, k.y, k.z );
        glm::vec3 sample = tbn * kernel_i;
        sample = sample * ssao_info.radius + frag.position_vs + frag.normal;
        
        glm::vec4 vox =voxel( volume, glm::floor( sample ) );
        occlusion += vox.a;
    }

    
    frag.color *= (1.f - occlusion / ssao_info.kernelSize );


    // Convert both the position and the normal to view-space
    // We export (some of) these values for use in later passes
    frag.position = glm::vec3( glm::mat3(raycast_info.matModelView) * ( frag.position - r.origin ) );
    frag.normal = glm::normalize( glm::mat3( raycast_info.matModelView ) * frag.normal );
    float depth =frag.position.z;

    
    // Write the color information to the framebuffer
    uint32_t rgba;
//    packRGBA_UINT32( &rgba, glm::vec4( frag.position.z, frag.position.z, frag.position.z, 1.f ) );
    packRGBA_UINT32( &rgba, frag.color  );

    // Workaround to be able to write to a 24bit buffer. Saves conversion later
    surf2Dwrite<uint8_t>( (uint8_t)( (rgba >> 24) & 0xFF), fb_color_surface, x*3, y, hipBoundaryModeTrap );
    surf2Dwrite<uint8_t>( (uint8_t)( (rgba >> 16) & 0xFF), fb_color_surface, x*3+1, y, hipBoundaryModeTrap );
    surf2Dwrite<uint8_t>( (uint8_t)( (rgba >> 8) & 0xFF), fb_color_surface, x*3+2, y, hipBoundaryModeTrap );


    // Write the normal and depth values using a regular 32 float4 texture
    float4 normal_depth;
    normal_depth.w =depth;
    normal_depth.x =frag.normal.x;
    normal_depth.y =frag.normal.y;
    normal_depth.z =frag.normal.z;

    surf2Dwrite<float4>( normal_depth, fb_normal_depth_surface, x * sizeof( float4 ), y, hipBoundaryModeTrap ); 

}

VOXOWL_CUDA_KERNEL
void
computeFragmentSSAO( raycastInfo_t raycast_info, ssaoInfo_t ssao_info, framebufferDevice_t framebuffer, glm::mat4 mat_projection ) {
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    float occlusion =0.f;

    // Read back the values from the previous pass
    uint8_t r =surf2Dread<uint8_t>( fb_color_surface, x*3, y, hipBoundaryModeTrap );
    uint8_t g =surf2Dread<uint8_t>( fb_color_surface, x*3+1, y, hipBoundaryModeTrap );
    uint8_t b =surf2Dread<uint8_t>( fb_color_surface, x*3+2, y, hipBoundaryModeTrap );
    
    // Obtain the normal and depth values for this fragment
    float4 normal_depth =surf2Dread<float4>( fb_normal_depth_surface, x * sizeof( float4 ), y, hipBoundaryModeTrap );
    glm::vec3 normal( normal_depth.x, normal_depth.y, normal_depth.z );
    
    // Calculate the origin (position) of the fragment in view space using the depth
    glm::vec3 origin;
    glm::vec3 leftNormal = raycast_info.upperLeftNormal;
    glm::vec3 rightNormal = raycast_info.upperRightNormal;

    leftNormal += raycast_info.leftNormalYDelta * (float)y;
    rightNormal += raycast_info.rightNormalYDelta * (float)y;
    origin = leftNormal;
    
    glm::vec3 normalXDelta = (rightNormal - leftNormal) * raycast_info.invWidth;
    origin +=normalXDelta * (float)x;

    // We extend the calculated ray vector along the z axis by the fragment's depth
    origin =origin / -origin.z * normal_depth.w;


    // Obtain a random number to rotate the sampple matrix
    float4 noise =tex1D( ssao_noise, (float)x*y );
    glm::vec3 rvec( noise.x, noise.y, noise.z );

    // Setup the TBN matrix
    glm::vec3 tangent = glm::normalize(rvec - normal * glm::dot(rvec, normal));
    glm::vec3 bitangent = glm::cross(normal, tangent);
    glm::mat3 tbn(tangent, bitangent, normal);

    // Obtain the samples
    for (int i = 0; i < ssao_info.kernelSize; ++i) {
        // get sample position:

        float4 k =tex1D( ssao_kernel, i );
        glm::vec3 kernel_i( k.x, k.y, k.z );
        glm::vec3 sample = tbn * kernel_i;
        sample = sample * ssao_info.radius + origin;

        // project sample position:
        glm::vec4 offset = mat_projection * glm::vec4( origin, 1.f );
        glm::vec2 screenpos = glm::vec2( offset ) / offset.w ;
        screenpos.y =-screenpos.y;
        screenpos = screenpos * 0.5f + glm::vec2(0.5f);
        screenpos *= glm::vec2( framebuffer.width, framebuffer.height );

        // get sample depth:
        float4 normal_depth =surf2Dread<float4>( fb_normal_depth_surface, 
            (int)screenpos.x * sizeof( float4 ), 
            (int)screenpos.y, 
            hipBoundaryModeClamp );
        float sampleDepth = normal_depth.w;

        // range check & accumulate:
        float rangeCheck= glm::abs(origin.z - sampleDepth) < ssao_info.radius ? 1.0 : 0.0;
        occlusion += (sampleDepth >= sample.z ? 1.0 : 0.0) * rangeCheck;

 //       if( r > 1 ) {
 //           printf( "sample depth %f sample %f %f %f origin %f %f %f screenpos %f %f normal %f %f %f\n", sampleDepth, sample.x, sample.y, sample.z, origin.x, origin.y, origin.z, screenpos.x, screenpos.y, normal.x, normal.y, normal.z );
    //    }
    }

    // Apply the result
    occlusion = 1.f;// - occlusion / ssao_info.kernelSize;


    glm::vec3 color( r, g, b );
    color /= glm::vec3( 255.f );
    color *= occlusion;
    
    surf2Dwrite<uint8_t>( color.r * 255.f, fb_color_surface, x*3, y, hipBoundaryModeTrap );
    surf2Dwrite<uint8_t>( color.g * 255.f, fb_color_surface, x*3+1, y, hipBoundaryModeTrap );
    surf2Dwrite<uint8_t>( color.b * 255.f, fb_color_surface, x*3+2, y, hipBoundaryModeTrap );
}

RaycasterCUDA::RaycasterCUDA( const char* name, Object* parent ) 
    : Renderer( name, parent ) {
    bzero( &d_volume, sizeof( volumeDevice_t ) );
    bzero( &d_framebuffer, sizeof( framebufferDevice_t ) );
    if (!initSSAO() ) {
        fprintf( stderr, "initSSAO(): %s\n", errorString().c_str() ); 
    }
}

RaycasterCUDA::~RaycasterCUDA() {

        if( d_framebuffer.color_data )
            hipFreeArray( d_framebuffer.color_data );
        if( d_framebuffer.normal_depth_data )
            hipFreeArray( d_framebuffer.normal_depth_data );
        if( d_volume.data )
            hipFreeArray( d_volume.data ); 
}

bool 
RaycasterCUDA::beginRender() {

    if( !getFramebuffer() )
        return setError( true, "No framebuffer set" );
    if( !getVolume() )
        return setError( true, "No input volume set" );
    if( !getCamera() )
        return setError( true, "No camera set" );

    const int width =getFramebuffer()->getWidth();
    const int height =getFramebuffer()->getHeight();
    const dim3 blocksize(16, 16);

    // For now, we only use a voxelmap as input
    voxelmap_t voxelmap =getVolume()->data();
    
    // Allocate the volume on the device, if neccesary
    bool realloc_volume = !d_volume.data || ( d_volume.size != glm_ivec3_16( voxelmap.size ) ) || ( d_volume.format != voxelmap.format );

    if( realloc_volume ) { // Reallocate the volume on the device end
        if( d_volume.data )
            RETURN_IF_ERR( hipFreeArray( d_volume.data ) );

        d_volume.size =glm_ivec3_16( voxelmap.size );
        d_volume.blocks =glm_ivec3_16( voxelmap.blocks );
        d_volume.format =voxelmap.format;

        hipExtent v_extent;
        hipChannelFormatDesc v_channelDesc;

        printf( "Allocating texture for voxelmap, blocks=(%d,%d,%d), bytes per block=%d\n", voxelmap.blocks.x, voxelmap.blocks.y, voxelmap.blocks.z, bytesPerBlock( voxelmap.format ) );

        switch( voxelmap.format ) {
            // We differentiate between byte and word block sizes
            case VOXEL_RGB24A1_UINT32:
            case VOXEL_RGB24_8ALPHA1_UINT32:
            case VOXEL_RGBA_UINT32:
                v_extent = make_hipExtent( d_volume.blocks.x, d_volume.blocks.y, d_volume.blocks.z );
                v_channelDesc = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindUnsigned);
                break;

            case VOXEL_INTENSITY_UINT8:
            case VOXEL_BITMAP_UINT8:
                // By convention, we store blocks per-byte in the z-direction
                v_extent = make_hipExtent( d_volume.blocks.x, d_volume.blocks.y, d_volume.blocks.z );
                v_channelDesc = hipCreateChannelDesc(8,0,0,0,hipChannelFormatKindUnsigned);
                break;

            default:
                return setError( true, "Unhandled voxel format" );
                
        }

        RETURN_IF_ERR( hipMalloc3DArray( &d_volume.data, &v_channelDesc, v_extent ) );
        volume_texture.normalized = false;                      
        volume_texture.filterMode = hipFilterModePoint;      
        volume_texture.addressMode[0] = hipAddressModeClamp;   
        volume_texture.addressMode[1] = hipAddressModeClamp;
        volume_texture.addressMode[2] = hipAddressModeClamp;
        RETURN_IF_ERR( hipBindTextureToArray( volume_texture, d_volume.data, v_channelDesc ) );
        
        // Copy the volume to the device
        // TODO: make separate step
        hipMemcpy3DParms copyParams = {0};
        copyParams.srcPtr   = make_hipPitchedPtr( voxelmap.data, v_extent.width * bytesPerBlock(voxelmap.format), v_extent.width, v_extent.height);
        copyParams.dstArray = d_volume.data;
        copyParams.extent   = v_extent;
        copyParams.kind     = hipMemcpyHostToDevice;
        RETURN_IF_ERR( hipMemcpy3D(&copyParams) );
    }
    
    // Allocate the framebuffer on the device, if neccesary
    bool realloc_framebuffer = !d_framebuffer.color_data 
        || ( d_framebuffer.width != width ) 
        || ( d_framebuffer.height != height )
        || ( (int)d_framebuffer.format != getFramebuffer()->getPixelFormat() );
    
    d_framebuffer.aaXSamples =getFramebuffer()->getAAXSamples();
    d_framebuffer.aaYSamples =getFramebuffer()->getAAYSamples();

    if( realloc_framebuffer ) { // Reallocate the framebuffer on the device end
        if( d_framebuffer.color_data )
            RETURN_IF_ERR( hipFreeArray( d_framebuffer.color_data ) );
        if( d_framebuffer.normal_depth_data )
            RETURN_IF_ERR( hipFreeArray( d_framebuffer.normal_depth_data ) );

        d_framebuffer.width = width;
        d_framebuffer.height = height;
        d_framebuffer.format =(voxowl_pixel_format_t)getFramebuffer()->getPixelFormat();

        int bytes_per_pixel;
        // TODO: this could use a nice function
        if( d_framebuffer.format == VOXOWL_PF_RGB888 )
            bytes_per_pixel =3;
        else
            return false;

        // We allocate one buffer for the color data and optionally another for depth/normal data
        hipChannelFormatDesc fb_channelDesc = hipCreateChannelDesc(8,0,0,0,hipChannelFormatKindUnsigned);
        RETURN_IF_ERR( hipMallocArray( &(d_framebuffer.color_data), &fb_channelDesc, width * bytes_per_pixel, height, hipArraySurfaceLoadStore ) );
        RETURN_IF_ERR( cudaBindSurfaceToArray( fb_color_surface, d_framebuffer.color_data ) );
        
        // We use a 32bit 4float type for the normal+depth buffer
        hipChannelFormatDesc fb_channelDesc2 = hipCreateChannelDesc(32,32,32,32,hipChannelFormatKindFloat);
        RETURN_IF_ERR( hipMallocArray( &(d_framebuffer.normal_depth_data), &fb_channelDesc2, width, height, hipArraySurfaceLoadStore ) );
        RETURN_IF_ERR( cudaBindSurfaceToArray( fb_normal_depth_surface, d_framebuffer.normal_depth_data ) );
    }

    
    // Setup the raycast parameters based on the matrices from the camera and the 'model'
    // TODO: some kind of caching?
    raycastInfo_t raycast_info;
    getCamera()->setAspect( (float)width/(float)height );
    raycastSetMatrices( &raycast_info, getVolume()->modelMatrix(), getCamera()->getViewMatrix(), getCamera()->getProjMatrix(), width, height );

    // Divide the invidual fragments over N / blocksize blocks
    // Run the raycast kernel on the device
    const dim3 numblocks( width / blocksize.x, height / blocksize.y );
    RETURN_IF_ERR( cudaBindSurfaceToArray( fb_color_surface, d_framebuffer.color_data ) );
    RETURN_IF_ERR( cudaBindSurfaceToArray( fb_normal_depth_surface, d_framebuffer.normal_depth_data ) );
    computeFragment<<<numblocks, blocksize>>>( raycast_info, d_volume, d_framebuffer, ssao_info, getCamera()->getProjMatrix() );
    RETURN_IF_ERR( hipGetLastError() );

    // Experimental SSAO step
    RETURN_IF_ERR( hipDeviceSynchronize() );
    computeFragmentSSAO<<<numblocks, blocksize>>>( raycast_info, ssao_info, d_framebuffer, getCamera()->getProjMatrix() );
    RETURN_IF_ERR( hipGetLastError() );

    return true;
}

bool 
RaycasterCUDA::synchronize() {
    // Wait for the running kernel to finish
    RETURN_IF_ERR( hipDeviceSynchronize() );

    // Copy the framebuffer to the host

    // TODO: use framebuffer format
    int bpp =3;
    void* data_ptr =getFramebuffer()->data();
    int width =getFramebuffer()->getWidth();
    int height =getFramebuffer()->getHeight();


    RETURN_IF_ERR ( hipMemcpyFromArray( data_ptr, d_framebuffer.color_data, 0, 0, width*height*bpp, hipMemcpyDeviceToHost ) );
    

    return true;
}

VOXOWL_HOST
bool
RaycasterCUDA::initSSAO() {
    static const int KERNEL_SIZE =512;
    static const int NOISE_SIZE =64;
    //static const float RADIUS =1.f;
    // for VSAO
    static const float RADIUS =20.0f;

    ssao_info.kernelSize =KERNEL_SIZE;
    ssao_info.noiseSize =NOISE_SIZE;
    ssao_info.radius =RADIUS;
    
    glm::vec4 kernel[ssao_info.kernelSize];
    glm::vec4 noise[ssao_info.noiseSize];


    for (int i = 0; i < ssao_info.kernelSize; ++i) {
        float scale = (float)i / (float)ssao_info.kernelSize;
        //scale = glm::mix(0.1f, 1.0f, scale * scale);
        // for VSAO
        scale = glm::mix(0.5f, 1.0f, scale * scale);
        
        kernel[i] = glm::normalize( glm::vec4 (
            random(-1.0f, 1.0f),
            random(-1.0f, 1.0f),
            random(0.0f, 1.0f),
            0.f ) );
//        kernel[i] *= random(0.1f, 1.0f);
        kernel[i] *= scale;
//        kernel[i].z += 1.f;
    }

    for (int i = 0; i < ssao_info.noiseSize; ++i) {
        noise[i] = glm::normalize( glm::vec4(
            random(-1.0f, 1.0f),
            random(-1.0f, 1.0f),
            0.0f, 0.0f  ) ) ;
    }

    hipChannelFormatDesc fb_channelDesc = hipCreateChannelDesc(32,32,32,32,hipChannelFormatKindFloat);
    RETURN_IF_ERR( hipMallocArray( &(ssao_info.noise), &fb_channelDesc, ssao_info.noiseSize, 1, 0 ) );
    ssao_noise.filterMode = hipFilterModeLinear;
    ssao_noise.normalized = false;                      
    ssao_noise.addressMode[0] = hipAddressModeWrap;   
    RETURN_IF_ERR( hipBindTextureToArray( ssao_noise, ssao_info.noise ) );
    RETURN_IF_ERR( hipMemcpyToArray( ssao_info.noise, 0, 0, noise, ssao_info.noiseSize * sizeof( float4 ), hipMemcpyHostToDevice ) );

    RETURN_IF_ERR( hipMallocArray( &(ssao_info.sampleKernel), &fb_channelDesc, ssao_info.kernelSize, 1, 0 ) );
    ssao_kernel.filterMode = hipFilterModeLinear;
    ssao_kernel.normalized = false;                      
    ssao_kernel.addressMode[0] = hipAddressModeWrap;   
    RETURN_IF_ERR( hipBindTextureToArray( ssao_kernel, ssao_info.sampleKernel ) );
    RETURN_IF_ERR( hipMemcpyToArray( ssao_info.sampleKernel, 0, 0, kernel, ssao_info.kernelSize * sizeof( float4 ), hipMemcpyHostToDevice ) );
    
    return true;
}
